#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C++" {
#include "glife.h"
}
#include <hip/hip_runtime.h>

#define BLOCK_NUM 10
#define THREAD_PER_B_NUM 10

// HINT: YOU CAN USE THIS METHOD FOR ERROR CHECKING
// Print error message on CUDA API or kernel launch
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err), \
                    __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
        } \
    } while (0)

__device__ int isLive_cu(int*grid, int index){
    return (grid[index] ? 1 : 0);
}

// TODO: YOU MAY NEED TO USE IT OR CREATE MORE
__device__ int getNeighbors(int* grid, int tot_rows, int tot_cols, int row, int col) {
    int numOfNeighbors = 0;
    int index = row*tot_cols + col;

    if(row>=1          && col>=1          && isLive_cu(grid,index -tot_cols -1)){numOfNeighbors ++;}
    if(row>=1          &&                    isLive_cu(grid,index -tot_cols   )){numOfNeighbors ++;}
    if(row>=1          && col<=tot_cols-2 && isLive_cu(grid,index -tot_cols +1)){numOfNeighbors ++;}
    if(                   col>=1          && isLive_cu(grid,index           -1)){numOfNeighbors ++;}
    if(                   col<=tot_cols-2 && isLive_cu(grid,index           +1)){numOfNeighbors ++;}
    if(row<=tot_rows-2 && col>=1          && isLive_cu(grid,index +tot_cols -1)){numOfNeighbors ++;}
    if(row<=tot_rows-2 &&                    isLive_cu(grid,index +tot_cols   )){numOfNeighbors ++;}
    if(row<=tot_rows-2 && col<=tot_cols-2 && isLive_cu(grid,index +tot_cols +1)){numOfNeighbors ++;}

    return numOfNeighbors;
}

// TODO: YOU NEED TO IMPLEMENT KERNEL TO RUN ON GPU DEVICE
__global__ void kernel(int * readGrid, int * writeGrid, int tot_rows, int tot_cols, int BN, int TpBN)
{
    int width; int margin;
    if(tot_rows*tot_cols > BN*TpBN){
        width = (tot_rows*tot_cols) / (BN*TpBN);
        margin = (tot_rows*tot_cols) % (BN*TpBN);
    }
    else{
        width = 1;
        margin = 0;
    }




    int from = blockIdx.x*TpBN*width + (threadIdx.x * width);
    int to = from + width -1;
    int max = tot_rows * tot_cols;

    int tindex = blockIdx.x * TpBN + threadIdx.x;
    if(BN*TpBN -1 -tindex < margin and BN*TpBN -1 -tindex >=0){
        int i = margin -BN*TpBN +1 +tindex;
        from = from + i;
        to = to + i;
    }

    while(from <=to && from<max){
        int i = from / tot_cols;
        int j = from % tot_cols;

        int nei = getNeighbors(readGrid, tot_rows, tot_cols, i, j);
        if( (isLive_cu(readGrid,from)&&nei==2) || nei==3) { writeGrid[from] = 1;}
        else                                              { writeGrid[from] = 0;}

        from ++;
    }
}




// TODO: YOU NEED TO IMPLEMENT TO PRINT THE INDEX RESULTS
void cuda_dump(GameOfLifeGrid * golg)
{
    golg->dump();
}

// TODO: YOU NEED TO IMPLEMENT TO PRINT THE INDEX RESULTS
void cuda_dump_index(GameOfLifeGrid * golg)
{
    golg->dumpIndex();
}

// TODO: YOU NEED TO IMPLEMENT ON CUDA VERSION
uint64_t runCUDA(int rows, int cols, int gen,
                 GameOfLifeGrid* g_GameOfLifeGrid, int display)
{
    hipSetDevice(0); // DO NOT CHANGE THIS LINE

    uint64_t difft;

    // ---------- TODO: CALL CUDA API HERE ----------
    int * Grid1;
    int * Grid2;
    hipMalloc((void **)&Grid1, sizeof(int) * rows * cols);
    hipMalloc((void **)&Grid2, sizeof(int) * rows * cols);

    int *cpuArr = g_GameOfLifeGrid->get1DGrid();

    hipMemcpy(Grid1, cpuArr, sizeof(int)*rows*cols, hipMemcpyHostToDevice);

    // Start timer for CUDA kernel execution
    difft = dtime_usec(0);
    // ----------  TODO: CALL KERNEL HERE  ----------
    int g= 0;
    while(g<gen){
        if(g%2==1) {kernel<<<BLOCK_NUM,THREAD_PER_B_NUM>>>(Grid2, Grid1, rows, cols,
                        BLOCK_NUM, THREAD_PER_B_NUM);} //odd num
        else    {kernel<<<BLOCK_NUM,THREAD_PER_B_NUM>>>(Grid1, Grid2, rows, cols,
                        BLOCK_NUM, THREAD_PER_B_NUM);} //even num
        g++;

    }

    // Finish timer for CUDA kernel execution
    int *newcpuArr;
    newcpuArr = (int*)malloc(sizeof(int)*rows*cols);
    if(g%2==1) {hipMemcpy(newcpuArr, Grid2, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);}
    else    {hipMemcpy(newcpuArr, Grid1, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);}


    g_GameOfLifeGrid->updateGrid(newcpuArr);

    difft = dtime_usec(difft);

   // Print the results
    if (display) {
        cuda_dump(g_GameOfLifeGrid);
        cuda_dump_index(g_GameOfLifeGrid);
    }

    hipFree(Grid1);
    hipFree(Grid2);
    return difft;
}
#endif
